#include "hip/hip_runtime.h"
/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <mpi.h>


#define MAXLINE 2000
#define MAXCAD 200

/*	Important assumption: each process will run on a different GPU. That is, no two (or more) processes will share
 *	the same GPU. In order to make the sharing possible, change the value here below accordingly:
 *	
 *		- 0: all processes will run on the same GPU
 *		- 1: all processes will run on different GPUs. Mind that the following must hold:
 *	
 * 				|processes| = |GPUs|
 *	
 **/

#define SINGLE_GPU_PER_PROCESS 0

//Macros
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */

#define CHECK_CUDA_CALL(a) { \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

#define CHECK_MPI_CALL(a) { \
    if (a != MPI_SUCCESS) { \
        fprintf(stderr, "[ERROR] Fatal error with MPI in line %d. Aborting\n", __LINE__); \
        MPI_Abort(MPI_COMM_WORLD, a); \
    } \
}

/* 
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char* filename)
{
	printf("Error\n");
	switch (error)
	{
		case -1:
			fprintf(stderr,"\tFile %s has too many columns.\n", filename);
			fprintf(stderr,"\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
			break;
		case -2:
			fprintf(stderr,"Error reading file: %s.\n", filename);
			break;
		case -3:
			fprintf(stderr,"Error writing file: %s.\n", filename);
			break;
	}
	fflush(stderr);	
}

/* 
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char* filename, int *lines, int *samples)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int contlines, contsamples = 0;
    
    contlines = 0;

    if ((fp=fopen(filename,"r"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL) 
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
            contlines++;       
            ptr = strtok(line, delim);
            contsamples = 0;
            while(ptr != NULL)
            {
            	contsamples++;
				ptr = strtok(NULL, delim);
	    	}	    
        }
        fclose(fp);
        *lines = contlines;
        *samples = contsamples;  
        return 0;
    }
    else
	{
    	return -2;
	}
}

/* 
Function readInput2: It loads data from file.
*/
int readInput2(char* filename, float* data)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int i = 0;
    
    if ((fp=fopen(filename,"rt"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
        {         
            ptr = strtok(line, delim);
            while(ptr != NULL)
            {
            	data[i] = atof(ptr);
            	i++;
				ptr = strtok(NULL, delim);
	   		}
	    }
        fclose(fp);
        return 0;
    }
    else
	{
    	return -2; //No file found
	}
}

/* 
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int *classMap, int lines, const char* filename)
{	
    FILE *fp;
    
    if ((fp=fopen(filename,"wt"))!=NULL)
    {
        for(int i=0; i<lines; i++)
        {
        	fprintf(fp,"%d\n",classMap[i]);
        }
        fclose(fp);  
   
        return 0;
    }
    else
	{
    	return -3; //No file found
	}
}

/*
Function initCentroids: This function copies the values of the initial centroids, using their 
position in the input data structure as a reference map.
*/
void initCentroids(const float *data, float* centroids, int* centroidPos, int samples, int K)
{
	int i;
	int idx;
	for(i = 0; i < K; i++) {
		idx = centroidPos[i];
		memcpy(&centroids[i * samples], &data[idx * samples], (samples * sizeof(float)));
	}
}

/*
Function euclideanDistance: Euclidean distance
This function could be modified
*/
__device__ void euclideanDistance(float *point, float *center, int samples, float* return_addr) {
	float dist = 0.0;
	for (int i = 0; i < samples; i++) {
		dist += (point[i] - center[i]) * (point[i] - center[i]);
	}
	*return_addr = sqrt(dist);
}

/*
Function zeroFloatMatriz: Set matrix elements to 0
This function could be modified
*/
void zeroFloatMatriz(float *matrix, int rows, int columns) {
	int i, j;
	for (i = 0; i < rows; i++) {
		for (j = 0; j < columns; j++) {
			matrix[i * columns + j] = 0.0;
		}
	}
}

/*
Function zeroIntArray: Set array elements to 0
This function could be modified
*/
void zeroIntArray(int *array, int size) {
	int i;
	for (i = 0; i < size; i++) {
		array[i] = 0;
	}
}


/* 
 *		CUDA Kernels and Variables
 */

__constant__ int gpu_K;
__constant__ int gpu_n;
__constant__ int gpu_d;

/*  To each thread, a point with D dimensions gets assigned. The thread must compute the
 *  l_2 norm and take the minimum. Then, for each such point, get the associated cluster,
 * 	and count the number of points for each cluster. Then, for each point, sum its
 * 	coordinates into a matrix which is used for doing the average of the coordinates.
 * 	After that, average all the coordinates and check the maximum distance that changed.
 *
 *  Parameters:
 * 		- `data`: array of points, on the GPU;
 * 		- `centroids`: array of centroids, on the GPU;
 * 		- `class_map`: array with the classes, on the GPU.
 * 		- `changes_return`: address to which the total changes should be written on;
 * 		- `centroids_table`: pointer to the table for storing the centroids dimensions, on the GPU;
 * 		- `points_per_class`: pointer to the table storing the amount of points for each class, on the GPU;
 * 
 *  Returns:
 * 		- `NULL`
 */
__global__ void step_1_kernel(float* data, float* centroids, int* points_per_class, float* centroids_table, int* class_map, int* changes_return) {
	// Compute thread index
	int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
							(threadIdx.y * blockDim.x) +
							threadIdx.x;

	extern __shared__ float shared_centroids[];	// K x D x sizeof(float)

	// Define block size and local thread index (index within block)
	int block_size = blockDim.x * blockDim.y;
	int local_thread_index = threadIdx.x + threadIdx.y * blockDim.x;

	// Copy centroids data into shared memory
	for (int portion = 0; portion < (gpu_K * gpu_d) / block_size; portion++) {
		int copy_index = local_thread_index + portion * block_size;
		shared_centroids[copy_index] = centroids[copy_index];
	}

	if (thread_index < gpu_n) {
		int data_index = thread_index * gpu_d;
		int class_int = class_map[thread_index];
		float min_dist = FLT_MAX;
		
		// For each centroid...
		for (int centroid = 0; centroid < gpu_K; centroid++) {
			float distance;

			// Compute the euclidean distance
			euclideanDistance(&data[data_index], &centroids[centroid * gpu_d], gpu_d, &distance);

			// If distance is smaller, replace the distance and assign new class
			if (distance < min_dist) {
				min_dist = distance;
				class_int = centroid + 1;
			}
		}

		// If the class is different, add one change and write new class
		if (class_map[thread_index] != class_int) {
			atomicAdd(changes_return, 1);
		}
		
		// Map the value to the class map
		class_map[thread_index] = class_int;

		int class_assignment = class_map[thread_index];
		int point_index = class_assignment - 1;

		// Atomically increase the number of points for the given class
		atomicAdd(&(points_per_class[point_index]), 1);

		for (int dim = 0; dim < gpu_d; dim++) {
			int index = point_index * gpu_d + dim;
			atomicAdd(&centroids_table[index], data[data_index + dim]);
		}
	}
}


/*  To each thread, assign a centroid. The coordinates of each centroid get averaged, and then the
 *  l_2 norm gets computed in order to check the max distance
 *  
 *  
 *  Parameters:
 * 		- `centroids_table`: a table with all the temporary new coordinates of the centroids, on the GPU;
 * 		- `centroids`: array with the centroids, on the GPU;
 * 		- `points_per_class`: a table enumerating how many points have been assigned for each class, on the GPU;
 * 		- `dimensions`: the number of dimensions of each point;
 *  
 *  Returns:
 * 		- `NULL`
 */
__global__ void step_2_kernel(float* centroids_table, float* centroids, int* points_per_class, float* max_distance) {
	// Index of the thread
	int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
							(threadIdx.y * blockDim.x) +
							threadIdx.x;
	
	if (thread_index < gpu_K) {
		float distance;
		for (int d = 0; d < gpu_d; d++) {
			centroids_table[thread_index * gpu_d + d] /= (float) points_per_class[thread_index];
			// Compute Euclidean distance (l_2 norm) to check for maximum distance
			distance += pow((centroids[thread_index * gpu_d + d] - centroids_table[thread_index * gpu_d + d]), 2);
		}

		// Perform sqrt of distance
		distance = sqrt(distance);

		if (distance > *max_distance) {
			// Exchange atomically, disregard old value
			atomicExch(max_distance, distance);
		}
	}
}


int main(int argc, char* argv[]) {

    //  ###################################
    //              MPI Section
    //  ###################################

    int init = MPI_Init(&argc, &argv);
    CHECK_MPI_CALL(init);

    int rank, comm_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
    MPI_Comm_set_errhandler(MPI_COMM_WORLD, MPI_ERRORS_RETURN);


	//START CLOCK***************************************
	#ifdef _OPEN_MP
		double start, end;
		start = omp_get_wtime();
	#else
		clock_t start, end;
		start = MPI_Wtime();
	#endif
	//**************************************************

	/*
	* PARAMETERS
	*
	* argv[1]: Input data file
	* argv[2]: Number of clusters
	* argv[3]: Maximum number of iterations of the method. Algorithm termination condition.
	* argv[4]: Minimum percentage of class changes. Algorithm termination condition.
	*          If between one iteration and the next, the percentage of class changes is less than
	*          this percentage, the algorithm stops.
	* argv[5]: Precision in the centroid distance after the update.
	*          It is an algorithm termination condition. If between one iteration of the algorithm 
	*          and the next, the maximum distance between centroids is less than this precision, the
	*          algorithm stops.
	* argv[6]: Output file. Class assigned to each point of the input file.
	* */

    // lines = number of points; samples = number of dimensions per point
    int lines = 0, samples= 0;

	float* data;
	float* centroids;

    void* partial_centroids;	// Will hold K / size items

    // If your rank is 0...
    if (rank == 0) {
        // ...initialize the data
        if (argc !=  7) {
            fprintf(stderr,"EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
            fprintf(stderr,"./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Number of changes] [Threshold] [Output data file]\n");
            fflush(stderr);
            exit(-1);
            MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
        }

        // Reading the input data
        int error = readInput(argv[1], &lines, &samples);
        if (error != 0) {
            showFileError(error,argv[1]);
            MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
            exit(error);
        }
        
        data = (float*) calloc(lines * samples, sizeof(float));
        if (data == NULL) {
            fprintf(stderr,"Memory allocation error.\n");
            MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
            exit(-4);
        }

        error = readInput2(argv[1], data);
        if(error != 0) {
            showFileError(error,argv[1]);
            MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
            exit(error);
        }
	}

	// Broadcast N and D
	MPI_Bcast(&lines, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&samples, 1, MPI_INT, 0, MPI_COMM_WORLD);
        
	// Parameters from the input args
	int K = atoi(argv[2]); 
	int maxIterations = atoi(argv[3]);
	int minChanges = (int) (lines * atof(argv[4]) / 100.0);
	float maxThreshold = atof(argv[5]);

	centroids = (float*) calloc(K * samples, sizeof(float));
	
	if (centroids == NULL) {
		fprintf(stderr, "Memory allocation error.\n");
		MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
		exit(-4);
	}

	if (rank == 0) {
		int* centroidPos = (int*) calloc(K, sizeof(int));
		//int* classMap = (int*) calloc(lines, sizeof(int));

		// Initial centrodis
		srand(0);
		int i;
		for (i = 0; i < K; i++) 
			centroidPos[i] = rand() % lines;
		
		// Loading the array of initial centroids with the data from the array data
		// The centroids are points stored in the data array.
		initCentroids(data, centroids, centroidPos, samples, K);
		free(centroidPos);

		printf("\n    Input properties:");
		printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], lines, samples);
		printf("\tNumber of clusters: %d\n", K);
		printf("\tMaximum number of iterations: %d\n", maxIterations);
		printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, atof(argv[4]), lines);
		printf("\tMaximum centroid precision: %f\n", maxThreshold);

		// Check CUDA device properties
		hipDeviceProp_t cuda_prop;

		CHECK_CUDA_CALL(hipGetDeviceProperties(&cuda_prop, 0));

		printf("\n    Device: %s\n", cuda_prop.name);
		printf("\tCompute Capability: %d.%d\n", cuda_prop.major, cuda_prop.minor);
		printf("\tMax threads / block: %d\n", cuda_prop.maxThreadsPerBlock);
		printf("\tMax threads / SM: %d\n", cuda_prop.maxThreadsPerMultiProcessor);
		printf("\tMax blocks / SM: %d\n", cuda_prop.maxBlocksPerMultiProcessor);
		printf("\tMax grid size: %d x %d x %d\n", cuda_prop.maxGridSize[0], cuda_prop.maxGridSize[1], cuda_prop.maxGridSize[2]);
		printf("\tMax shared memory per SM: %dB\n", cuda_prop.sharedMemPerMultiprocessor);
		printf("\tNumber of SMs: %d\n", cuda_prop.multiProcessorCount);
		printf("\tStarting with following grids and blocks:\n");
		printf("\t    Blocks: 32 x 32\n");
		printf("\t    Grid for points: %d x (32 x 32)\n", lines / (32 * 32) + 1);
		printf("\t    Grid for centroids: %d x (32 x 32)\n", K / (32 * 32) + 1);
	}

	int local_n = lines / comm_size;
	float* partial_data = (float*) calloc(local_n * samples, sizeof(float));
	int *pointsPerClass = (int*) malloc(K * sizeof(int)); 
	float *auxCentroids = (float*) malloc(K * samples * sizeof(float)); 

	MPI_Bcast(centroids, K * samples, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Scatter(data, lines * samples, MPI_FLOAT, partial_data, local_n * samples, MPI_FLOAT, 0, MPI_COMM_WORLD);	


	
	//END CLOCK*****************************************
	#ifdef _OPEN_MP
		end = omp_get_wtime();
	#else
		end = MPI_Wtime();
	#endif
	printf("\nMemory allocation: %f seconds\n", end - start);
	fflush(stdout);

	// Check assumption on top of file for better explanation on this part

	#if SINGLE_GPU_PER_PROCESS == 0 
		CHECK_CUDA_CALL( hipSetDevice(0) );
	#elif SINGLE_GPU_PER_PROCESS == 1
		CHECK_CUDA_CALL( hipSetDevice(rank) );
	#else
		printf("Invalid SINGLE_GPU_PER_PROCESS value. Aborting\n");
		MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
		exit(-4);
	#endif

	CHECK_CUDA_CALL( hipDeviceSynchronize() );

	//**************************************************
	//START CLOCK***************************************
	#ifdef _OPEN_MP
		start = omp_get_wtime();
	#else
		start = MPI_Wtime();
	#endif
	//**************************************************
	char *outputMsg = (char *)calloc(100000,sizeof(char));
	char line[100];

	int it = 0;
	int changes = 0;
	float maxDist = FLT_MIN;

	//pointPerClass: number of points classified in each class
	//auxCentroids: mean of the points in each class
	//int *local_pointsPerClass = (int*) malloc(K / comm_size * sizeof(int));
	//float *local_auxCentroids = (float*) malloc((K * samples) * sizeof(float));
	//float *distCentroids = (float*) malloc(K * sizeof(float)); 
	//if (local_pointsPerClass == NULL || local_auxCentroids == NULL || distCentroids == NULL) {
	//	fprintf(stderr,"Memory allocation error.\n");
	//	exit(-4);
	//}

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

    //  ###################################
    //              CUDA Section
    //  ###################################

	// Set carveout to be of maximum size available
	int carveout = cudaSharedmemCarveoutMaxShared;

	CHECK_CUDA_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(step_1_kernel), hipFuncAttributePreferredSharedMemoryCarveout, carveout));

	// CUDA related variables
	int pts_grid_size = (lines / comm_size) / (32 * 32) + 1;
	int K_grid_size = (K / comm_size) / (32 * 32) + 1;

	dim3 gen_block(32, 32);
	dim3 dyn_grid_pts(pts_grid_size);
	dim3 dyn_grid_cent(K_grid_size);

	int data_size = lines * samples * sizeof(float);
	int centroids_size = K * samples * sizeof(float);

	// GPU pointers
	float* gpu_data;
	float* gpu_centroids;
	int* gpu_class_map;
	float* gpu_centroids_temp;
	int* gpu_points_per_class;

	// Loop-iteration needed vars
	int* gpu_changes;
	float* gpu_max_distance;

	// Load data into the GPU

	/*	Variables:
	 *		- centroids: same for all;
	 *		- data: split;
	 *		- class_map: split;
	 *		- centroids_temp: split;
	 * 		- points_per_class: split;
	 * 		- changes: split;
	 * 		- max_distance: split;
	 * 
	 * 
	 *  */


	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_centroids, centroids_size));
	CHECK_CUDA_CALL(hipMemcpy(gpu_centroids, centroids, centroids_size, hipMemcpyHostToDevice));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_data, data_size / comm_size));
	CHECK_CUDA_CALL(hipMemcpy(gpu_data, partial_data, data_size / comm_size, hipMemcpyHostToDevice));
	
	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_class_map, lines / comm_size * sizeof(int)));
	CHECK_CUDA_CALL(hipMemset(gpu_class_map, 0, lines / comm_size * sizeof(int)));
	
	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_centroids_temp, centroids_size / comm_size));
	CHECK_CUDA_CALL(hipMemset(gpu_centroids_temp, 0, centroids_size / comm_size));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_points_per_class, K / comm_size * sizeof(int)));
	CHECK_CUDA_CALL(hipMemset(gpu_points_per_class, 0, K / comm_size * sizeof(int)));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_changes, sizeof(int)));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_max_distance, sizeof(float)));

	// Initialize constant vars
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_K), &K, sizeof(int)));
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_n), &lines, sizeof(int)));
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_d), &samples, sizeof(int)));

	do {
		it++;

		// Reset changes, max distance, table of centroids and points per class
		CHECK_CUDA_CALL(hipMemset(gpu_changes, 0, sizeof(int)));
		CHECK_CUDA_CALL(hipMemset(gpu_max_distance, FLT_MIN, sizeof(float)));
		CHECK_CUDA_CALL(hipMemset(gpu_centroids_temp, 0, centroids_size / comm_size * sizeof(int)));
		CHECK_CUDA_CALL(hipMemset(gpu_points_per_class, 0, K / comm_size * sizeof(int)));
		
		// Ensure memory is actually ready for being used
		CHECK_CUDA_CALL(hipDeviceSynchronize());

		// 1. Calculate the distance from each point to the centroid

		// Assign each point to the nearest centroid.
		step_1_kernel<<<dyn_grid_pts, gen_block, centroids_size / comm_size>>>(gpu_data, gpu_centroids, gpu_points_per_class,
			gpu_centroids_temp, gpu_class_map, gpu_changes);
		CHECK_CUDA_LAST();

		int* local_changes;
		int* local_points_per_class;
		MPI_Request MPI_changes_reduce_handler;

		CHECK_CUDA_CALL(hipMemcpy(local_changes, gpu_changes, sizeof(int), hipMemcpyDeviceToHost));
		CHECK_CUDA_CALL(hipMemcpy(pointsPerClass, gpu_points_per_class, K * sizeof(int), hipMemcpyDeviceToHost));
		CHECK_CUDA_CALL(hipMemcpy(auxCentroids, gpu_centroids_temp, K * samples * sizeof(float), hipMemcpyDeviceToHost));
		
		// Write down to host the changes for checking convergence condition after waiting for GPU
		CHECK_CUDA_CALL(hipDeviceSynchronize());
		
		MPI_Ireduce(local_changes, &changes, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD, &MPI_changes_reduce_handler);
		MPI_Allreduce(MPI_IN_PLACE, pointsPerClass, K, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
		MPI_Allreduce(MPI_IN_PLACE, auxCentroids, K * samples, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);

		// 2. Recalculates the centroids: calculates the mean within each cluster
        
		// Perform the second update step, on the centroids
		step_2_kernel<<<dyn_grid_cent, gen_block>>>(gpu_centroids_temp, gpu_centroids, gpu_points_per_class, gpu_max_distance);
		CHECK_CUDA_LAST();

		// Update effectively the positions and take maxDist
		CHECK_CUDA_CALL(hipMemcpy(&maxDist, gpu_max_distance, sizeof(float), hipMemcpyDeviceToHost));
		CHECK_CUDA_CALL(hipMemcpy(gpu_centroids, gpu_centroids_temp, centroids_size, hipMemcpyDeviceToDevice));
		
		CHECK_CUDA_CALL(hipDeviceSynchronize());

		sprintf(line,"\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, maxDist);
		outputMsg = strcat(outputMsg, line);

	} while((changes > minChanges) && (it < maxIterations) && (maxDist > maxThreshold));

	// Needed after loop
	int* partial_class_map = (int*) calloc(local_n, sizeof(int));

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */
	// Output and termination conditions
	printf("%s",outputMsg);	

	CHECK_CUDA_CALL( hipDeviceSynchronize() );

	//END CLOCK*****************************************
	#ifdef _OPEN_MP
		end = omp_get_wtime();
	#else
		end = MPI_Wtime();
	#endif
	printf("\nComputation: %f seconds", end - start);
	fflush(stdout);
	//**************************************************
	//START CLOCK***************************************
	#ifdef _OPEN_MP
		start = omp_get_wtime();
	#else
		start = MPI_Wtime();
	#endif
	//**************************************************

	if (rank == 0) {
		int* classMap;




		if (changes <= minChanges) {
			printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
		}
		else if (it >= maxIterations) {
			printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
		}
		else {
			printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
		}	

		// Writing the classification of each point to the output file.
		CHECK_CUDA_CALL(hipMemcpy(classMap, gpu_class_map, lines * sizeof(int), hipMemcpyDeviceToHost));
		CHECK_CUDA_CALL(hipDeviceSynchronize());

		int error = writeResult(classMap, lines, argv[6]);
		if(error != 0)
		{
			showFileError(error, argv[6]);
			exit(error);
		}

		//Free memory
		free(data);
		free(classMap);
	}	

	free(partial_data);
	free(centroids);

	hipFree(gpu_data);
	hipFree(gpu_centroids);
	hipFree(gpu_centroids_temp);
	hipFree(gpu_changes);
	hipFree(gpu_class_map);
	hipFree(gpu_max_distance);
	hipFree(gpu_points_per_class);

	//END CLOCK*****************************************
	#ifdef _OPEN_MP
		end = omp_get_wtime();
	#else
		end = MPI_Wtime();
	#endif
	printf("\n\nMemory deallocation: %f seconds\n", end - start);
	fflush(stdout);
	//***************************************************/
    MPI_Finalize();
	return 0;
}