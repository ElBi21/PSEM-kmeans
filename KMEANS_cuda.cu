/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <hip/hip_runtime.h>


#define MAXLINE 2000
#define MAXCAD 200

//Macros
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL(a) { \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

/* 
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char* filename)
{
	printf("Error\n");
	switch (error)
	{
		case -1:
			fprintf(stderr,"\tFile %s has too many columns.\n", filename);
			fprintf(stderr,"\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
			break;
		case -2:
			fprintf(stderr,"Error reading file: %s.\n", filename);
			break;
		case -3:
			fprintf(stderr,"Error writing file: %s.\n", filename);
			break;
	}
	fflush(stderr);	
}

/* 
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char* filename, int *lines, int *samples)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int contlines, contsamples = 0;
    
    contlines = 0;

    if ((fp=fopen(filename,"r"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL) 
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
            contlines++;       
            ptr = strtok(line, delim);
            contsamples = 0;
            while(ptr != NULL)
            {
            	contsamples++;
				ptr = strtok(NULL, delim);
	    	}	    
        }
        fclose(fp);
        *lines = contlines;
        *samples = contsamples;  
        return 0;
    }
    else
	{
    	return -2;
	}
}

/* 
Function readInput2: It loads data from file.
*/
int readInput2(char* filename, float* data)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int i = 0;
    
    if ((fp=fopen(filename,"rt"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
        {         
            ptr = strtok(line, delim);
            while(ptr != NULL)
            {
            	data[i] = atof(ptr);
            	i++;
				ptr = strtok(NULL, delim);
	   		}
	    }
        fclose(fp);
        return 0;
    }
    else
	{
    	return -2; //No file found
	}
}

/* 
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int *classMap, int lines, const char* filename)
{	
    FILE *fp;
    
    if ((fp=fopen(filename,"wt"))!=NULL)
    {
        for(int i=0; i<lines; i++)
        {
        	fprintf(fp,"%d\n",classMap[i]);
        }
        fclose(fp);  
   
        return 0;
    }
    else
	{
    	return -3; //No file found
	}
}

/*
Function initCentroids: This function copies the values of the initial centroids, using their 
position in the input data structure as a reference map.
*/
void initCentroids(const float *data, float* centroids, int* centroidPos, int samples, int K)
{
	int i;
	int idx;
	for(i = 0; i < K; i++) {
		idx = centroidPos[i];
		memcpy(&centroids[i * samples], &data[idx * samples], (samples * sizeof(float)));
	}
}

/*
Function euclideanDistance: Euclidean distance
This function could be modified
*/
__device__ void euclideanDistance(float *point, float *center, int samples, float* return_addr) {
	float dist = 0.0;
	for (int i = 0; i < samples; i++) {
		dist += (point[i] - center[i]) * (point[i] - center[i]);
	}
	*return_addr = dist;
}

/*
Function zeroFloatMatriz: Set matrix elements to 0
This function could be modified
*/
void zeroFloatMatriz(float *matrix, int rows, int columns) {
	int i, j;
	for (i = 0; i < rows; i++) {
		for (j = 0; j < columns; j++) {
			matrix[i * columns + j] = 0.0;
		}
	}
}

/*
Function zeroIntArray: Set array elements to 0
This function could be modified
*/
void zeroIntArray(int *array, int size) {
	int i;
	for (i = 0; i < size; i++) {
		array[i] = 0;
	}
}


/* 
 *		CUDA Kernels and Variables
 */

__constant__ int gpu_K;
__constant__ int gpu_n;
__constant__ int gpu_d;


/*	Implementation of a custom atomicMax operation for floats. Freely taken from
 *	https://stackoverflow.com/questions/17399119. Credits to vinograd47
 *  (User link: https://stackoverflow.com/users/2451683/vinograd47)
 */
__device__ float custom_atomic_max(float* value_address, float val) {
    int* address_as_int = (int*) value_address;
    int old = *address_as_int, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}


/*  To each thread, a point with D dimensions gets assigned. The thread must compute the l_2 norm and
 *  take the minimum.
 *
 *  Parameters:
 * 		- `data`: array of points, on the GPU;
 * 		- `centroids`: array of centroids, on the GPU;
 * 		- `class_map`: array with the classes, on the GPU.
 * 		- `changes_return`: address to which the total changes should be written on;
 * 
 *  Returns:
 * 		- `NULL`
 */
__global__ void step_1_kernel(float* data, float* centroids, int* points_per_class, float* aux_centroids, int* class_map, int* changes_return) {
	// Compute thread index
	int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
							(threadIdx.y * blockDim.x) +
							threadIdx.x;

	//extern __shared__ float shared_centroids[];	// K x D x sizeof(float)

	// Define block size and local thread index (index within block)
	//int block_size = blockDim.x * blockDim.y;
	//int local_thread_index = threadIdx.x + threadIdx.y * blockDim.x;

	// Copy centroids data into shared memory
	//for (int portion = 0; portion < (gpu_K * gpu_d) / block_size; portion++) {
	//	int copy_index = local_thread_index + portion * block_size;
	//	shared_centroids[copy_index] = centroids[copy_index];
	//}

	if (thread_index < gpu_n) {
		int data_index = thread_index * gpu_d;
		int class_int = class_map[thread_index];
		float min_dist = FLT_MAX;
		
		// For each centroid...
		for (int centroid = 0; centroid < gpu_K; centroid++) {
			float distance = 0.0f;

			// Compute the euclidean distance
			euclideanDistance(&data[data_index], &centroids[centroid * gpu_d], gpu_d, &distance);

			// If distance is smaller, replace the distance and assign new class
			if (distance < min_dist) {
				min_dist = distance;
				class_int = centroid + 1;
			}
		}

		// If the class is different, add one change and write new class
		if (class_map[thread_index] != class_int) {
			atomicAdd(changes_return, 1);
		}
		
		// Map the value to the class map
		class_map[thread_index] = class_int;

		int class_assignment = class_map[thread_index];
		int point_index = class_assignment - 1;

		atomicAdd(&(points_per_class[point_index]), 1);

		//if (thread_index == 0)
		//	printf("  PPC: %d\n", points_per_class[thread_index]);

		for (int dim = 0; dim < gpu_d; dim++) {
			int index = point_index * gpu_d + dim;
			atomicAdd(&aux_centroids[index], data[data_index + dim]);
		}
	}
}


/*  To each thread, assign a centroid. The coordinates of each centroid get averaged, and then the
 *  l_2 norm gets computed in order to check the max distance
 *  
 *  
 *  Parameters:
 * 		- `centroids_table`: a table with all the temporary new coordinates of the centroids, on the GPU;
 * 		- `centroids`: array with the centroids, on the GPU;
 * 		- `points_per_class`: a table enumerating how many points have been assigned for each class, on the GPU;
 * 		- `dimensions`: the number of dimensions of each point;
 *  
 *  Returns:
 * 		- `NULL`
 */
__global__ void step_2_kernel(float* centroids_table, float* centroids, int* points_per_class, float* max_distance) {
	// Index of the thread
	int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
							(threadIdx.y * blockDim.x) +
							threadIdx.x;
	
	if (thread_index < gpu_K) {
		float distance;
		for (int d = 0; d < gpu_d; d++) {
			centroids_table[thread_index * gpu_d + d] /= (float) points_per_class[thread_index];
			// Compute Euclidean distance (l_2 norm) to check for maximum distance
			distance += pow((centroids[thread_index * gpu_d + d] - centroids_table[thread_index * gpu_d + d]), 2);
		}

		// Perform sqrt of distance
		//distance = sqrt(distance);

		// Exchange atomically, disregard old value
		custom_atomic_max(max_distance, distance);
	}
}


int main(int argc, char* argv[])
{

	//START CLOCK***************************************
	clock_t start, end;
	start = clock();
	//**************************************************
	/*
	* PARAMETERS
	*
	* argv[1]: Input data file
	* argv[2]: Number of clusters
	* argv[3]: Maximum number of iterations of the method. Algorithm termination condition.
	* argv[4]: Minimum percentage of class changes. Algorithm termination condition.
	*          If between one iteration and the next, the percentage of class changes is less than
	*          this percentage, the algorithm stops.
	* argv[5]: Precision in the centroid distance after the update.
	*          It is an algorithm termination condition. If between one iteration of the algorithm 
	*          and the next, the maximum distance between centroids is less than this precision, the
	*          algorithm stops.
	* argv[6]: Output file. Class assigned to each point of the input file.
	* */
	if(argc !=  7)
	{
		fprintf(stderr,"EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
		fprintf(stderr,"./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Number of changes] [Threshold] [Output data file]\n");
		fflush(stderr);
		exit(-1);
	}

	// Reading the input data
	// lines = number of points; samples = number of dimensions per point
	int n = 0, d = 0;  
	
	int error = readInput(argv[1], &n, &d);
	if(error != 0) {
		showFileError(error,argv[1]);
		exit(error);
	}
	
	float *data = (float*) calloc(n * d, sizeof(float));
	if (data == NULL) {
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}
	
	error = readInput2(argv[1], data);
	if(error != 0) {
		showFileError(error,argv[1]);
		exit(error);
	}

	// Parameters
	int K = atoi(argv[2]); 
	int max_iterations = atoi(argv[3]);
	int min_changes = (int) (n * atof(argv[4]) / 100.0);
	int max_threshold = pow(atof(argv[5]), 2);

	int* centroid_pos = (int*) calloc(K, sizeof(int));
	float* centroids = (float*) calloc(K * d, sizeof(float));
	int* class_map = (int*) calloc(n, sizeof(int));

    if (centroid_pos == NULL || centroids == NULL || class_map == NULL)
	{
		fprintf(stderr, "Memory allocation error.\n");
		exit(-4);
	}

	// Initial centrodis
	srand(0);
	int i;
	for(i = 0; i < K; i++) 
		centroid_pos[i] = rand() % n;
	
	// Loading the array of initial centroids with the data from the array data
	// The centroids are points stored in the data array.
	initCentroids(data, centroids, centroid_pos, d, K);


	printf("\n    Input properties:");
	printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], n, d);
	printf("\tNumber of clusters: %d\n", K);
	printf("\tMaximum number of iterations: %d\n", max_iterations);
	printf("\tMinimum number of changes: %d [%g%% of %d points]\n", min_changes, atof(argv[4]), n);
	printf("\tMaximum centroid precision: %f\n", max_threshold);


	// Check CUDA device properties
	hipDeviceProp_t cuda_prop;

	// Adapt to the number of points
	int pts_grid_size = n / (32 * 32) + 1;
	int K_grid_size = K / (32 * 32) + 1;

	CHECK_CUDA_CALL(hipGetDeviceProperties(&cuda_prop, 0));

	printf("\n    Device: %s\n", cuda_prop.name);
	printf("\tCompute Capability: %d.%d\n", cuda_prop.major, cuda_prop.minor);
	printf("\tMax threads / block: %d\n", cuda_prop.maxThreadsPerBlock);
	printf("\tMax threads / SM: %d\n", cuda_prop.maxThreadsPerMultiProcessor);
	printf("\tMax blocks / SM: %d\n", cuda_prop.maxBlocksPerMultiProcessor);
	printf("\tMax grid size: %d x %d x %d\n", cuda_prop.maxGridSize[0], cuda_prop.maxGridSize[1], cuda_prop.maxGridSize[2]);
	printf("\tMax shared memory per SM: %dB\n", cuda_prop.sharedMemPerMultiprocessor);
	printf("\tNumber of SMs: %d\n", cuda_prop.multiProcessorCount);
	printf("\tStarting with following grids and blocks:\n");
	printf("\t    Blocks: 32 x 32\n");
	printf("\t    Grid for points: %d x (32 x 32)\n", pts_grid_size);
	printf("\t    Grid for centroids: %d x (32 x 32)\n", K_grid_size);


	
	//END CLOCK*****************************************
	end = clock();
	printf("\nMemory allocation: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
	fflush(stdout);

	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	//**************************************************
	//START CLOCK***************************************
	start = clock();
	//**************************************************
	char *output_msg = (char*) calloc(100000, sizeof(char));
	char line[100];

	int it = 0;
	int changes = 0;
	float max_dist = FLT_MIN;

	//pointPerClass: number of points classified in each class
	//auxCentroids: mean of the points in each class
	int* points_per_class = (int*) malloc(K * sizeof(int));
	float* aux_centroids = (float*) malloc(K * d * sizeof(float));
	float* dist_centroids = (float*) malloc(K * sizeof(float)); 
	if (points_per_class == NULL || aux_centroids == NULL || dist_centroids == NULL) {
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}


	// Set carveout to be of maximum size available
	int carveout = cudaSharedmemCarveoutMaxShared;

	CHECK_CUDA_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(step_1_kernel), hipFuncAttributePreferredSharedMemoryCarveout, carveout));

	dim3 gen_block(32, 32);
	dim3 dyn_grid_pts(pts_grid_size);
	dim3 dyn_grid_cent(K_grid_size);

	int data_size = n * d * sizeof(float);
	int centroids_size = K * d * sizeof(float);

	// GPU pointers
	float* gpu_data;
	float* gpu_centroids;
	int* gpu_class_map;
	float* gpu_aux_centroids;
	int* gpu_points_per_class;

	// Loop-iteration needed vars
	int* gpu_changes;
	float* gpu_max_distance;

	// Load data into the GPU
	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_centroids, centroids_size));
	CHECK_CUDA_CALL(hipMemcpy(gpu_centroids, centroids, centroids_size, hipMemcpyHostToDevice));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_data, data_size));
	CHECK_CUDA_CALL(hipMemcpy(gpu_data, data, data_size, hipMemcpyHostToDevice));
	
	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_class_map, n * sizeof(int)));
	CHECK_CUDA_CALL(hipMemset(gpu_class_map, 0, n * sizeof(int)));
	
	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_aux_centroids, centroids_size));
	CHECK_CUDA_CALL(hipMemset(gpu_aux_centroids, 0, centroids_size));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_points_per_class, K * sizeof(int)));
	CHECK_CUDA_CALL(hipMemset(gpu_points_per_class, 0, K * sizeof(int)));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_changes, sizeof(int)));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_max_distance, sizeof(float)));

	// Initialize constant vars
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_K), &K, sizeof(int)));
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_n), &n, sizeof(int)));
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_d), &d, sizeof(int)));

	//END CLOCK*****************************************
	end = clock();
	printf("\nCUDA initialization: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
	fflush(stdout);
	//**************************************************
	//START CLOCK***************************************
	start = clock();
	//**************************************************

	do {
		it++;

		// Reset changes, max distance, table of centroids and points per class
		CHECK_CUDA_CALL(hipMemset(gpu_changes, 0, sizeof(int)));
		CHECK_CUDA_CALL(hipMemset(gpu_max_distance, FLT_MIN, sizeof(float)));
		CHECK_CUDA_CALL(hipMemset(gpu_aux_centroids, 0, K * d * sizeof(int)));
		CHECK_CUDA_CALL(hipMemset(gpu_points_per_class, 0, K * sizeof(int)));
		
		// Ensure memory is actually ready for being used
		CHECK_CUDA_CALL(hipDeviceSynchronize());

		// 1. Calculate the distance from each point to the centroid

		// Assign each point to the nearest centroid.
		step_1_kernel<<<dyn_grid_pts, gen_block, centroids_size>>>(gpu_data, gpu_centroids, gpu_points_per_class,
			gpu_aux_centroids, gpu_class_map, gpu_changes);
		CHECK_CUDA_LAST();

		// Write down to host the changes for checking convergence condition after waiting for GPU
		CHECK_CUDA_CALL(hipDeviceSynchronize());
		CHECK_CUDA_CALL(hipMemcpy(&changes, gpu_changes, sizeof(int), hipMemcpyDeviceToHost));

		// 2. Recalculates the centroids: calculates the mean within each cluster
        
		// Perform the second update step, on the centroids
		step_2_kernel<<<dyn_grid_cent, gen_block>>>(gpu_aux_centroids, gpu_centroids, gpu_points_per_class, gpu_max_distance);
		CHECK_CUDA_LAST();

		// Update effectively the positions and take maxDist
		CHECK_CUDA_CALL(hipMemcpy(&max_dist, gpu_max_distance, sizeof(float), hipMemcpyDeviceToHost));
		CHECK_CUDA_CALL(hipMemcpy(gpu_centroids, gpu_aux_centroids, centroids_size, hipMemcpyDeviceToDevice));
		
		CHECK_CUDA_CALL(hipDeviceSynchronize());

		sprintf(line,"\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, max_dist);
		output_msg = strcat(output_msg, line);

	} while((changes > min_changes) && (it < max_iterations) && (max_dist > pow(max_threshold, 2)));


/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */
	// Output and termination conditions
	printf("%s", output_msg);	

	CHECK_CUDA_CALL( hipDeviceSynchronize() );

	//END CLOCK*****************************************
	end = clock();
	printf("\nComputation: %f seconds", (double)(end - start) / CLOCKS_PER_SEC);
	fflush(stdout);
	//**************************************************
	//START CLOCK***************************************
	start = clock();
	//**************************************************

	

	if (changes <= min_changes) {
		printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, min_changes);
	}
	else if (it >= max_iterations) {
		printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, max_iterations);
	}
	else {
		printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", max_dist, max_threshold);
	}	

	// Writing the classification of each point to the output file.
	CHECK_CUDA_CALL(hipMemcpy(class_map, gpu_class_map, n * sizeof(int), hipMemcpyDeviceToHost));
	CHECK_CUDA_CALL(hipDeviceSynchronize());

	error = writeResult(class_map, n, argv[6]);
	if(error != 0)
	{
		showFileError(error, argv[6]);
		exit(error);
	}

	//Free memory
	free(data);
	free(class_map);
	free(centroid_pos);
	free(centroids);
	free(dist_centroids);
	free(points_per_class);
	free(aux_centroids);

	hipFree(gpu_data);
	hipFree(gpu_centroids);
	hipFree(gpu_aux_centroids);
	hipFree(gpu_changes);
	hipFree(gpu_class_map);
	hipFree(gpu_max_distance);
	hipFree(gpu_points_per_class);

	//END CLOCK*****************************************
	end = clock();
	printf("\n\nMemory deallocation: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
	fflush(stdout);
	//***************************************************/
	return 0;
}