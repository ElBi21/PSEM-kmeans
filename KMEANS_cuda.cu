/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <hip/hip_runtime.h>


#define MAXLINE 2000
#define MAXCAD 200

//Macros
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL(a) { \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

/*
 * 		CUDA Constants
 */
__constant__ int gpu_K;
__constant__ int gpu_n;
__constant__ int gpu_d;


/*
 *	===	Program Functions ===
 */

 
/* 
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char* filename)
{
	printf("Error\n");
	switch (error)
	{
		case -1:
			fprintf(stderr,"\tFile %s has too many columns.\n", filename);
			fprintf(stderr,"\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
			break;
		case -2:
			fprintf(stderr,"Error reading file: %s.\n", filename);
			break;
		case -3:
			fprintf(stderr,"Error writing file: %s.\n", filename);
			break;
	}
	fflush(stderr);	
}

/* 
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char* filename, int *lines, int *samples)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int contlines, contsamples = 0;
    
    contlines = 0;

    if ((fp=fopen(filename,"r"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL) 
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
            contlines++;       
            ptr = strtok(line, delim);
            contsamples = 0;
            while(ptr != NULL)
            {
            	contsamples++;
				ptr = strtok(NULL, delim);
	    	}	    
        }
        fclose(fp);
        *lines = contlines;
        *samples = contsamples;  
        return 0;
    }
    else
	{
    	return -2;
	}
}

/* 
Function readInput2: It loads data from file.
*/
int readInput2(char* filename, float* data)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int i = 0;
    
    if ((fp=fopen(filename,"rt"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
        {         
            ptr = strtok(line, delim);
            while(ptr != NULL)
            {
            	data[i] = atof(ptr);
            	i++;
				ptr = strtok(NULL, delim);
	   		}
	    }
        fclose(fp);
        return 0;
    }
    else
	{
    	return -2; //No file found
	}
}

/* 
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int* classMap, int lines, const char* filename) {	
    FILE *fp;
    
    if ((fp = fopen(filename, "wt")) != NULL) {
        for (int i = 0; i < lines; i++) {
        	fprintf(fp,"%d\n", classMap[i]);
        }

        fclose(fp);  
   
        return 0;
    } else {
    	return -3; // No file found
	}
}

/*
Function initCentroids: This function copies the values of the initial centroids, using their 
position in the input data structure as a reference map.
*/
void initCentroids(const float *data, float* centroids, int* centroidPos, int samples, int K)
{
	int i;
	int idx;
	for(i = 0; i < K; i++) {
		idx = centroidPos[i];
		memcpy(&centroids[i * samples], &data[idx * samples], (samples * sizeof(float)));
	}
}

/*
Function euclideanDistance: Euclidean distance
This function could be modified
*/
__device__ void euclideanDistance(float* data, float* centroids, int data_idx, int centroid_idx, float* return_addr) {
	float dist = 0.0;
	for (int dim = 0; dim < gpu_d; dim++) {
		float temp = (data[data_idx + gpu_n * dim] - centroids[centroid_idx * gpu_d + dim]);
		dist += temp * temp;
	}

	*return_addr = dist;
}

/*
Function zeroFloatMatriz: Set matrix elements to 0
This function could be modified
*/
void zeroFloatMatriz(float *matrix, int rows, int columns) {
	int i, j;
	for (i = 0; i < rows; i++) {
		for (j = 0; j < columns; j++) {
			matrix[i * columns + j] = 0.0;
		}
	}
}

/*
Function zeroIntArray: Set array elements to 0
This function could be modified
*/
void zeroIntArray(int *array, int size) {
	int i;
	for (i = 0; i < size; i++) {
		array[i] = 0;
	}
}

/* The function will transpose the array passed as input, so that to make it coalescent for CUDA */
float* transpose(float *array, int N, int D) {
    float* coalescedArray = (float*) malloc(N * D * sizeof(float));

    if (coalescedArray == NULL) {
        fprintf(stderr, "Memory allocation error.\n");
        exit(-4);
    }

    for (int n = 0; n < N; n++) {
        for (int d = 0; d < D; d++) {
            coalescedArray[d * N + n] = array[n * D + d];
        }
    }
    
	return coalescedArray;
}

/*	Implementation of a custom atomicMax operation for floats. Freely taken from
 *	https://stackoverflow.com/questions/17399119. Credits to vinograd47
 *  (User link: https://stackoverflow.com/users/2451683/vinograd47)
 */
__device__ float custom_atomicMax(float* value_address, float val) {
    int* address_as_int = (int*) value_address;
    int old = *address_as_int, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

/* 
 *	===	CUDA Kernels ===
 */

/*  To each thread, a point with D dimensions gets assigned. The thread must compute the l_2 norm and
 *  take the minimum.
 *
 *  Parameters:_m
 * 		- `data`: array of points, on the GPU;
 * 		- `centroids`: array of centroids, on the GPU;
 * 		- `class_map`: array with the classes, on the GPU.
 * 		- `changes_return`: address to which the total changes should be written on;
 * 
 *  Returns:
 * 		- `NULL`
 */
__global__ void assignment_step(float* data, float* centroids, int* class_map, int* changes_return) {
	// Compute thread index
	int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
						(threadIdx.y * blockDim.x) +
						threadIdx.x;

	extern __shared__ float shared_centroids[];	// K x D x sizeof(float)
	float distance = 0.0;

	// Define block size and local thread index (index within block)
	int block_size = blockDim.x * blockDim.y;
	int local_thread_index = threadIdx.x + threadIdx.y * blockDim.x;

	int total = gpu_K * gpu_d;
	for (int i = local_thread_index; i < total; i += block_size) {
 	   shared_centroids[i] = centroids[i];
	}

	// Wait until all centroids have been copied
	__syncthreads();

	if (thread_index < gpu_n) {
		// Here thread_index becomes the index of the data (since for each thread -> a point)
		int class_int = 1;
		float min_dist = FLT_MAX;
		
		// For each centroid...
		for (int centroid = 0; centroid < gpu_K; centroid++) {

			// Compute the euclidean distance
			euclideanDistance(data, shared_centroids, thread_index, centroid, &distance);

			// If distance is smaller, replace the distance and assign new class
			if (distance < min_dist) {
				min_dist = distance;
				class_int = centroid + 1;
			}
		}

		// If the class is different, add one change and write new class
		if (class_map[thread_index] != class_int) {
			atomicAdd(changes_return, 1);
		}
		
		// Map the value to the class map
		class_map[thread_index] = class_int;
	}
}


/*  To each thread, assign a point. For each such point, get the associated cluster,
 * 	and count the number of points for each cluster. Then, for each point, sum its
 * 	coordinates into a matrix which is used for doing the average of the coordinates.
 * 
 * 	Parameters:
 * 		- `data`: pointer to the data points, on the GPU;
 * 		- `class_map`: pointer to the cluster assignments, on the GPU;
 * 		- `aux_centroids`: pointer to the array for storing the new centroids dimensions, on the GPU;
 * 		- `points_per_class`: pointer to the table storing the amount of points for each class, on the GPU;
 * 
 *  Returns:
 * 		- `NULL`
 */
__global__ void update_step_points(float* data, int* class_map, float* aux_centroids, int* points_per_class) {
	// Index of the thread
	int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
						(threadIdx.y * blockDim.x) +
						threadIdx.x;

	extern __shared__ char shared_mem[];
	float* shared_aux_centroids = (float*) shared_mem;
    int* shared_points_per_class = (int*) (shared_mem + sizeof(float) * gpu_K * gpu_d);

	int block_size = blockDim.x * blockDim.y;
	int local_thread_index = threadIdx.x + threadIdx.y * blockDim.x;

	int total = gpu_K * gpu_d;
	for (int i = local_thread_index; i < total; i += block_size) {
 	   shared_aux_centroids[i] = 0.0;
	}

	for (int k = local_thread_index; k < gpu_K; k += block_size) {
        shared_points_per_class[k] = 0;
    }

    __syncthreads();

	if (thread_index < gpu_n) {
		// Get the class assignment for a given point
		int class_assignment = class_map[thread_index];
		int centroid_index = class_assignment - 1;

		// Add 1 to PPC in the corresponding centroid
		atomicAdd(&(shared_points_per_class[centroid_index]), 1);

		for (int dim = 0; dim < gpu_d; dim++) {
			// For each dimension, add to aux_centroids the coordinates
			atomicAdd(&(shared_aux_centroids[centroid_index * gpu_d + dim]), data[thread_index + gpu_n * dim]);
		}
	}

	__syncthreads();

	// Bring back the arrays to the global memory
	for (int k = local_thread_index; k < gpu_K; k += block_size) {
        atomicAdd(&points_per_class[k], shared_points_per_class[k]);
    }

	for (int i = local_thread_index; i < total; i += block_size) {
 		atomicAdd(&aux_centroids[i], shared_aux_centroids[i]) ;
	}

}

/*  To each thread, assign a centroid. The coordinates of each centroid get averaged, and then the
 *  l_2 norm gets computed in order to check the max distance
 *  
 *  
 *  Parameters:
 * 		- `aux_centroids`: an array with all the temporary new coordinates of the centroids, on the GPU;
 * 		- `centroids`: array with the centroids, on the GPU;
 * 		- `points_per_class`: a table enumerating how many points have been assigned for each class, on the GPU;
 * 		- `max_distance`: the pointer to the maxDistance value, which is needed for convergence;
 *  
 *  Returns:
 * 		- `NULL`
 */
__global__ void update_step_centroids(float* aux_centroids, float* centroids, int* points_per_class, float* max_distance) {
	// Index of the thread
	int thread_index = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + (blockIdx.x * blockDim.x * blockDim.y) +
						(threadIdx.y * blockDim.x) +
						threadIdx.x;


	int local_thread_index = threadIdx.x + threadIdx.y * blockDim.x;
	extern __shared__ float shared_max_distance[];
	if (local_thread_index == 0){
		shared_max_distance[0] = FLT_MIN;
	}

	// Eventually, make it run such that each thread is a dimensions,
	// the dimensions get averaged and then each thread does the distance or whatever
	if (thread_index < gpu_K) {
		float distance = 0.0;

		for (int dim = 0; dim < gpu_d; dim++) {
			aux_centroids[thread_index * gpu_d + dim] /= (float) points_per_class[thread_index];
			// Compute Euclidean distance (l_2 norm) to check for maximum distance
			distance += \
				(centroids[thread_index * gpu_d + dim] - aux_centroids[thread_index * gpu_d + dim]) * \
				(centroids[thread_index * gpu_d + dim] - aux_centroids[thread_index * gpu_d + dim]);

			// Update centroids within GPU
			centroids[thread_index * gpu_d + dim] = aux_centroids[thread_index * gpu_d + dim];
		}

		if (distance > shared_max_distance[0]) {
			// Atomic Max, disregard old value
			custom_atomicMax(&shared_max_distance[0], distance);
		}
	}

	__syncthreads();

	if (local_thread_index == 0){
		// Save the maxDistance on the GPU through a custom atomicMax for floats
		custom_atomicMax(max_distance, shared_max_distance[0]);
	}

}


int main(int argc, char* argv[]) {
	//START CLOCK***************************************
	clock_t start, end;
	start = clock();
	//**************************************************
	/*
	* PARAMETERS
	*
	* argv[1]: Input data file
	* argv[2]: Number of clusters
	* argv[3]: Maximum number of iterations of the method. Algorithm termination condition.
	* argv[4]: Minimum percfentage of class changes. Algorithm termination condition.
	*          If between one iteration and the next, the percentage of class changes is less than
	*          this percentage, the algorithm stops.
	* argv[5]: Precision in the centroid distance after the update.
	*          It is an algorithm termination condition. If between one iteration of the algorithm 
	*          and the next, the maximum distance between centroids is less than this precision, the
	*          algorithm stops.
	* argv[6]: Output file. Class assigned to each point of the input file.
	* */
	if (argc != 7) {
		fprintf(stderr,"EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
		fprintf(stderr,"./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Number of changes] [Threshold] [Output data file]\n");
		fflush(stderr);
		exit(-1);
	}

	// Reading the input data
	// lines = number of points; samples = number of dimensions per point
	int lines = 0, D= 0;  
	
	int error = readInput(argv[1], &lines, &D);
	if (error != 0) {
		showFileError(error,argv[1]);
		exit(error);
	}
	
	float *data = (float*) calloc(lines * D, sizeof(float));
	if (data == NULL) {
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

	error = readInput2(argv[1], data);
	if (error != 0) {
		showFileError(error,argv[1]);
		exit(error);
	}

	// Parameters
	int K = atoi(argv[2]); 
	int maxIterations = atoi(argv[3]);
	int minChanges = (int) (lines * atof(argv[4]) / 100.0);
	float maxThreshold = atof(argv[5]);

	int *centroidPos = (int*) calloc(K, sizeof(int));
	float *centroids = (float*) calloc(K * D, sizeof(float));
	int *classMap = (int*) calloc(lines, sizeof(int));

    if (centroidPos == NULL || centroids == NULL || classMap == NULL) {
		fprintf(stderr, "Memory allocation error.\n");
		exit(-4);
	}

	// Initial centrodis
	srand(0);
	int i;
	for (i = 0; i < K; i++) 
		centroidPos[i] = rand() % lines;
	
	// Loading the array of initial centroids with the data from the array data
	// The centroids are points stored in the data array.
	initCentroids(data, centroids, centroidPos, D, K);

	float* coalesced_data = transpose(data, lines, D);
	free(data);


	printf("\n    Input properties:");
	printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], lines, D);
	printf("\tNumber of clusters: %d\n", K);
	printf("\tMaximum number of iterations: %d\n", maxIterations);
	printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, atof(argv[4]), lines);
	printf("\tMaximum centroid precision: %f\n", maxThreshold);


	// Check CUDA device properties
	hipDeviceProp_t cuda_prop;

	// Adapt to the number of points
	int pts_grid_size = lines / (32 * 32) + 1;
	int K_grid_size = K / (32 * 32) + 1;

	CHECK_CUDA_CALL(hipGetDeviceProperties(&cuda_prop, 0));

	printf("\n    Device: %s\n", cuda_prop.name);
	printf("\tCompute Capability: %d.%d\n", cuda_prop.major, cuda_prop.minor);
	printf("\tMax threads / block: %d\n", cuda_prop.maxThreadsPerBlock);
	printf("\tMax threads / SM: %d\n", cuda_prop.maxThreadsPerMultiProcessor);
	printf("\tMax blocks / SM: %d\n", cuda_prop.maxBlocksPerMultiProcessor);
	printf("\tMax grid size: %d x %d x %d\n", cuda_prop.maxGridSize[0], cuda_prop.maxGridSize[1], cuda_prop.maxGridSize[2]);
	printf("\tMax shared memory per SM: %zuB\n", cuda_prop.sharedMemPerMultiprocessor);
	printf("\tNumber of SMs: %d\n", cuda_prop.multiProcessorCount);
	printf("\tStarting with following grids and blocks:\n");
	printf("\t    Blocks: 32 x 32\n");
	printf("\t    Grid for points: %d x (32 x 32)\n", pts_grid_size);
	printf("\t    Grid for centroids: %d x (32 x 32)\n", K_grid_size);


	// Output buffer allocation
	char *outputMsg = (char *)calloc(100000,sizeof(char));
	char line[200];

	int it = 0;
	int changes = 0;
	float maxDist = FLT_MIN;

	// pointPerClass: number of points classified in each class
	int *pointsPerClass = (int *) malloc(K * sizeof(int));
	
	// auxCentroids: mean of the points in each class
	float *auxCentroids = (float*) malloc(K * D * sizeof(float));
	
	if (pointsPerClass == NULL || auxCentroids == NULL) {
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}
	
	//END CLOCK*****************************************
	end = clock();
	printf("\nMemory allocation: %f seconds\n", (double) (end - start) / CLOCKS_PER_SEC);
	fflush(stdout);

	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	//**************************************************
	//START CLOCK***************************************
	start = clock();
	//**************************************************

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

	dim3 gen_block(32, 32);
	dim3 points_grid_size(pts_grid_size);
	dim3 centroids_grid_size(K_grid_size);

	int data_size = lines * D * sizeof(float);
	int centroids_size = K * D * sizeof(float);
	int pointperclass_size = K * sizeof(int);

	// GPU pointers
	float* gpu_data;
	float* gpu_centroids;
	int* gpu_class_map;
	float* gpu_aux_centroids;
	int* gpu_points_per_class;

	// Loop-iteration needed vars
	int* gpu_changes;
	float* gpu_max_distance;

	// Load data into the GPU
	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_centroids, centroids_size));
	CHECK_CUDA_CALL(hipMemcpy(gpu_centroids, centroids, centroids_size, hipMemcpyHostToDevice));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_data, data_size));
	CHECK_CUDA_CALL(hipMemcpy(gpu_data, coalesced_data, data_size, hipMemcpyHostToDevice));
	
	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_class_map, lines * sizeof(int)));
	CHECK_CUDA_CALL(hipMemset(gpu_class_map, 0, lines * sizeof(int)));
	
	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_aux_centroids, centroids_size));
	CHECK_CUDA_CALL(hipMemset(gpu_aux_centroids, 0, centroids_size));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_points_per_class, K * sizeof(int)));
	CHECK_CUDA_CALL(hipMemset(gpu_points_per_class, 0, K * sizeof(int)));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_changes, sizeof(int)));

	CHECK_CUDA_CALL(hipMalloc((void**) &gpu_max_distance, sizeof(float)));

	// Initialize constant vars
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_K), &K, sizeof(int)));
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_n), &lines, sizeof(int)));
	CHECK_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gpu_d), &D, sizeof(int)));

	//END CLOCK*****************************************
	end = clock();
	printf("\nCUDA Allocation: %f seconds\n", (double) (end - start) / CLOCKS_PER_SEC);
	fflush(stdout);
	//**************************************************
	//START CLOCK***************************************
	start = clock();
	//**************************************************

	do {
		it++;
		
		// Reset changes and max distance0.0
		CHECK_CUDA_CALL(hipMemset(gpu_changes, 0, sizeof(int)));
		CHECK_CUDA_CALL(hipMemset(gpu_max_distance, FLT_MIN, sizeof(float)));
		
		// 1. Calculate the distance from each point to the centroid
		// Assign each point to the nearest centroid.
		assignment_step<<<points_grid_size, gen_block, centroids_size>>>(gpu_data, gpu_centroids, gpu_class_map, gpu_changes);
		
		// Write down to host the changes for checking convergence condition after waiting for GPU
		CHECK_CUDA_CALL(hipDeviceSynchronize());
		CHECK_CUDA_CALL(hipMemcpy(&changes, gpu_changes, sizeof(int), hipMemcpyDeviceToHost));

		// 2. Recalculates the centroids: calculates the mean within each cluster
		// Perform the first update step, on the points
		update_step_points<<<points_grid_size, gen_block, centroids_size + pointperclass_size>>>(gpu_data, gpu_class_map, gpu_aux_centroids, gpu_points_per_class);
		CHECK_CUDA_LAST();

		CHECK_CUDA_CALL(hipDeviceSynchronize());

		// Perform the second update step, on the centroids
		update_step_centroids<<<centroids_grid_size, gen_block, sizeof(float)>>>(gpu_aux_centroids, gpu_centroids, gpu_points_per_class, gpu_max_distance);
		CHECK_CUDA_LAST();

		CHECK_CUDA_CALL(hipDeviceSynchronize());

		// Update effectively the positions and take maxDist
		CHECK_CUDA_CALL(hipMemcpy(&maxDist, gpu_max_distance, sizeof(float), hipMemcpyDeviceToHost));
		CHECK_CUDA_CALL(hipMemset(gpu_aux_centroids, 0.0, K * D * sizeof(float)));
		CHECK_CUDA_CALL(hipMemset(gpu_points_per_class, 0, K * sizeof(int)));

		CHECK_CUDA_CALL(hipDeviceSynchronize());
		
		sprintf(line, "\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, maxDist);
		outputMsg = strcat(outputMsg,line);
	} while ((changes > minChanges) && (it < maxIterations) && \
			(maxDist > pow(maxThreshold, 2)));


/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */
	// Output and termination conditions
	printf("%s",outputMsg);	

	CHECK_CUDA_CALL( hipDeviceSynchronize() );

	//END CLOCK*****************************************
	end = clock();
	printf("\nComputation: %f seconds", (double) (end - start) / CLOCKS_PER_SEC);
	fflush(stdout);
	//**************************************************
	//START CLOCK***************************************
	start = clock();
	//**************************************************



	if (changes <= minChanges) {
		printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
	} else if (it >= maxIterations) {
		printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
	} else {
		printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
	}	

	// Writing the classification of each point to the output file.
	CHECK_CUDA_CALL(hipMemcpy(classMap, gpu_class_map, lines * sizeof(int), hipMemcpyDeviceToHost));
	CHECK_CUDA_CALL(hipDeviceSynchronize());

	error = writeResult(classMap, lines, argv[6]);
	if(error != 0) {
		showFileError(error, argv[6]);
		exit(error);
	}

	//Free memory
	free(coalesced_data);
	free(classMap);
	free(centroidPos);
	free(centroids);
	free(pointsPerClass);
	free(auxCentroids);

	hipFree(gpu_data);
	hipFree(gpu_centroids);
	hipFree(gpu_aux_centroids);
	hipFree(gpu_changes);
	hipFree(gpu_class_map);
	hipFree(gpu_max_distance);
	hipFree(gpu_points_per_class);

	//END CLOCK*****************************************
	end = clock();
	printf("\n\nMemory deallocation: %f seconds\n", (double) (end - start) / CLOCKS_PER_SEC);
	fflush(stdout);
	//***************************************************/
	return 0;
}